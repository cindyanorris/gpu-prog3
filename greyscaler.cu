#include "hip/hip_runtime.h"
#include <sys/stat.h>
#include <stdlib.h>
#include <stdio.h>
#include <jpeglib.h>
#include <jerror.h>
#include "wrappers.h"
#include "h_colorToGreyscale.h"
#include "d_colorToGreyscale.h"

#define CHANNELS 3

//prototypes for functions in this file 
void checkCommandArgs(int, char **);
void printUsage();
void readJPGImage(char *, unsigned char **, int *, int *);
void writeJPGImage(char *, unsigned char *, int, int);
char * buildFilename(char *, const char *);
void compare(unsigned char * d_Pout, unsigned char * h_Pout, int size);

/*
    main 
    Opens the jpg file and reads the contents.  Uses the CPU
    and the GPU to perform the greyscale.  Compares the CPU and GPU
    results.  Writes the results to output files.  Outputs the
    time of each.
*/
int main(int argc, char * argv[])
{
    unsigned char * Pin;
    int width, height;
    checkCommandArgs(argc, argv);
    readJPGImage(argv[1], &Pin, &width, &height);

    //use the CPU to perform the greyscale
    unsigned char * h_Pout; 
    h_Pout = (unsigned char *) Malloc(sizeof(unsigned char) * width * height);
    float cpuTime = h_colorToGreyscale(h_Pout, Pin, width, height);
    char * h_outfile = buildFilename(argv[1], "h_grey");
    writeJPGImage(h_outfile, h_Pout, width, height);

    //use the GPU to perform the greyscale 
    unsigned char * d_Pout; 
    d_Pout = (unsigned char *) Malloc((sizeof(unsigned char) * width * height));
    float gpuTime = d_colorToGreyscale(d_Pout, Pin, width, height);
    char * d_outfile = buildFilename(argv[1], "d_grey");
    writeJPGImage(d_outfile, d_Pout, width, height);

    //compare the CPU and GPU results
    compare(d_Pout, h_Pout, width * height);

    printf("CPU time: %f msec\n", cpuTime);
    printf("GPU time: %f msec\n", gpuTime);
    printf("Speedup: %f\n", cpuTime/gpuTime);
    return EXIT_SUCCESS;
}

/* 
    compare
    This function takes two arrays of greyscale pixel values.  One array
    contains pixel values calculated  by the GPU.  The other array contains
    greyscale pixel values calculated by the CPU.  This function checks to
    see that the values are the same within a slight margin of error.

    d_Pout - pixel values calculated by GPU
    h_Pout - pixel values calculated by CPU
    size - size in elements of both arrays
    
    Outputs an error message and exits program if the arrays differ.
*/
void compare(unsigned char * d_Pout, unsigned char * h_Pout, int size)
{
    int i;
    for (i = 0; i < size; i++)
    {
        //GPU and CPU have different floating point standards so
        //the results could be slightly different
        int diff = d_Pout[i] - h_Pout[i];
        if (abs(diff) > 1)
        {
            printf("Greyscale results don't match.\n");
            printf("CPU pixel %d: %d\n", i, h_Pout[i]);
            printf("GPU pixel %d: %d\n", i, d_Pout[i]);
            exit(EXIT_FAILURE);
        }
    }
}

/* 
    writeJPGImage
    Writes a greyscale jpg image to an output file.

    outfile - name of jpg file (ends with a .jpg extension)
    Pout - array of pixels
    width - width (x-dimension) of image
    height - height (y-dimension) of image
*/
void writeJPGImage(char * fileName, unsigned char * Pout, int width, int height)
{
   struct jpeg_compress_struct cinfo;
   struct jpeg_error_mgr jerr;
   JSAMPROW rowPointer[1];

   //set up error handling
   cinfo.err = jpeg_std_error(&jerr);
   //initialize the compression object
   jpeg_create_compress(&cinfo);

   //open the output file
   FILE * fp;
   if ((fp = fopen(fileName, "wb")) == NULL)
   {
     fprintf(stderr, "Can't open %s\n", fileName);
     exit(1);
   }
   //initalize state for output to outfile
   jpeg_stdio_dest(&cinfo, fp);

   cinfo.image_width = width;    /* image width and height, in pixels */
   cinfo.image_height = height;
   cinfo.input_components = 1;   /* # of color components per pixel */
   cinfo.in_color_space = JCS_GRAYSCALE;
   jpeg_set_defaults(&cinfo);
   jpeg_set_quality(&cinfo, 75, TRUE);

   //TRUE means it will wrtie a complete interchange-JPEG file
   jpeg_start_compress(&cinfo, TRUE);

   while (cinfo.next_scanline < cinfo.image_height)
   {
      rowPointer[0] = &Pout[cinfo.next_scanline * width];
      (void) jpeg_write_scanlines(&cinfo, rowPointer, 1);
   }
   jpeg_finish_compress(&cinfo);
   fclose(fp);
   jpeg_destroy_compress(&cinfo);
}

/*
    buildFilename
    This function returns the concatenation of two strings by
    first allocating enough space to hold both strings and then
    copying the two strings into the allocated space.  
    It is used by the program to build the output file names.
*/    
char * buildFilename(char * infile, const char * prefix)
{
   int len = strlen(infile) + strlen(prefix) + 1;
   char * outfile = (char *) Malloc(sizeof(char *) * len);
   strncpy(outfile, prefix, strlen(prefix));
   strncpy(&outfile[strlen(prefix)], infile, strlen(infile) + 1);
   return outfile;
}
   
/*
    readJPGImage
    This function opens a jpg file and reads the contents.  

    Each pixel consists of bytes for red, green, and blue.  
    The array Pin is initialized to the pixel bytes.  width, height,
    to ints that are set to those values.
    filename - name of the .jpg file
*/
void readJPGImage(char * filename, unsigned char ** Pin, 
                  int * width, int * height)
{
   unsigned long dataSize;             // length of the file
   int channels;                       //  3 =>RGB   4 =>RGBA 
   unsigned char * rowptr[1];          // pointer to an array
   unsigned char * jdata;              // data for the image
   struct jpeg_decompress_struct info; //for our jpeg info
   struct jpeg_error_mgr err;          //the error handler

   FILE * fp = fopen(filename, "rb"); //read binary
   if (fp == NULL) 
   {
      fprintf(stderr, "Error reading file %s\n", filename);
      printUsage();
   }

   info.err = jpeg_std_error(& err);
   jpeg_create_decompress(&info);  

   jpeg_stdio_src(&info, fp);
   jpeg_read_header(&info, TRUE);   // read jpeg file header
   jpeg_start_decompress(&info);    // decompress the file

   //set width and height
   (*width) = info.output_width;
   (*height) = info.output_height;
   channels = info.num_components;
   if (channels != CHANNELS)
   {
      fprintf(stderr, "%s is not an RGB jpeg image\n", filename);
      printUsage();
   }

   dataSize = (*width) * (*height) * channels;
   jdata = (unsigned char *)Malloc(dataSize);
   while (info.output_scanline < info.output_height) // loop
   {
      // Enable jpeg_read_scanlines() to fill our jdata array
      rowptr[0] = (unsigned char *)jdata +  // secret to method
                  channels * info.output_width * info.output_scanline;

      jpeg_read_scanlines(&info, rowptr, 1);
   }
   jpeg_finish_decompress(&info);   //finish decompressing
   jpeg_destroy_decompress(&info);
   fclose(fp);                      //close the file
   (*Pin) = jdata;
   return;
}

/*
    checkCommandArgs
    This function checks the command line arguments.  Specifically,
    argv[1] should be the name of a .jpg file that exists.
*/
void checkCommandArgs(int argc, char * argv[])
{
    struct stat buffer;
    if (argc != 2) printUsage();

    int len = strlen(argv[1]);
    if (len < 5) printUsage();
    if (strncmp(".jpg", &argv[1][len - 4], 4) != 0) printUsage();

    //stat function returns 1 if file does not exist
    if (stat(argv[1], &buffer)) printUsage();
}

/*
    printUsage
    This function is called if there is an error in the command line
    arguments or if the .jpg file that is provided by the command line
    argument is improperly formatted.  It prints usage information and
    exits.
*/
void printUsage()
{
    printf("This application takes as input the name of a .jpg\n");
    printf("file containing a color image and creates a file\n");
    printf("containing a greyscale version of the file.\n");
    printf("\nusage: greyscaler <name>.jpg\n");
    exit(EXIT_FAILURE);
}
