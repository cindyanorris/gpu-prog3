#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "d_colorToGreyscale.h"
#include "CHECK.h"

#define CHANNELS 3
__global__ void d_colorToGreyscaleKernel(unsigned char *, unsigned char *,
                                         int, int);
/*
   d_colorToGreyscale
   Performs the greyscale of an image on the GPU.
   Pout array is filled with the greyscale of each pixel.
   Pin array contains the color pixels.
   width and height are the dimensions of the image.
*/
float d_colorToGreyscale(unsigned char * Pout, unsigned char * Pin,
                        int width, int height)
{
    hipEvent_t start_cpu, stop_cpu;
    float cpuMsecTime = -1;

    //Use cuda functions to do the timing 
    //create event objects
    CHECK(hipEventCreate(&start_cpu));
    CHECK(hipEventCreate(&stop_cpu));
    //record the starting time
    CHECK(hipEventRecord(start_cpu));

    //Your work goes here and in the kernel below

    //record the ending time and wait for event to complete
    CHECK(hipEventRecord(stop_cpu));
    CHECK(hipEventSynchronize(stop_cpu));
    //calculate the elapsed time between the two events 
    CHECK(hipEventElapsedTime(&cpuMsecTime, start_cpu, stop_cpu));
    return cpuMsecTime;
}

/*
   d_colorToGreyscaleKernel
   Kernel code executed by each thread on its own data when the kernel is
   launched.
   Pout array is filled with the greyscale of each pixel (one element per thread).
   Pin array contains the color pixels.
   width and height are the dimensions of the image.
*/
__global__
void d_colorToGreyscaleKernel(unsigned char * Pin, unsigned char * Pout,
                              int width, int height)
{

}
